#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// #include "cuda/simulat-kernel.cu"


int main ( void ) {
  mapRasterToAddresses(0,0);
  /*
    1. Initializing memory for raster records
    2. Read the CSV to host memory 
    3. Read the address records to host memory
    4. Map the addresses to a buffer host memory
    5. Shuffle raster and address buffer by a prime number
       each to the final host memory spaces
    6. copy host memory to device memory
  */

  return 0;
}