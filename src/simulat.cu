#include "hip/hip_runtime.h"
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "cuda/simulat-kernel.cu"

// Latitude Differenz = 0,002246  
// Longitude Differenz = 0,003255
int main ( void ) {
  for (globalThreadId = 0; globalThreadId < 3; globalThreadId++ ) {
    // mapRasterToAddresses(2000,4000);
  }
  int N = 1<<27;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  printf("%lu\n",sizeof(N));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
    /*
    1. Initializing memory for raster records
    2. Read the CSV to host memory 
    3. Read the address records to host memory
    4. Map the addresses to a buffer host memory
    5. Shuffle raster and address buffer by a prime number
       each to the final host memory spaces
    6. copy host memory to device memory
  */

  return 0;
}