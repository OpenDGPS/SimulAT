
#include <hip/hip_runtime.h>
#define NUMOFRASTERRECORDSPERCORE 160  // defined by num of raster records ~80k divided by num of GPU cores ~512
#define SIZEOFRASTERRECORD 4 // DWORDS to jump between the records

#define NUMOFADDRESSRECORDSPERCORE 5000 // defined by num of address records ~2.5m divided by num of GPU cores ~512
#define NUMOFADDRESSBLOCKS 512 // equal to the number of GPU cores
#define SIZEOFADDRESSRECORD 5 // DWORDS to jump between the records

__global__ void mapRasterToAddresses(int rasterRecords, int addressRecords) {

	int threadId, recordNum, addressBlockNum, currentAddressBlockNum, addressNumInBlock, rasterBase, addressBase, addressNum, currentRaster, currentAddress;
	
	for ( addressBlockNum = 0; addressBlockNum < NUMOFADDRESSBLOCKS; addressBlockNum++ ) {
	
		currentAddressBlockNum = ( addressBlockNum + threadId ) % NUMOFADDRESSBLOCKS;
	    
	    addressBase = addressRecords + ( currentAddressBlockNum * NUMOFADDRESSRECORDSPERCORE * SIZEOFADDRESSRECORD );

	    for ( recordNum = 0; recordNum < NUMOFRASTERRECORDSPERCORE; recordNum++ ) {

	    	currentRaster = rasterRecords + ( recordNum * SIZEOFRASTERRECORD ) + ( threadId * SIZEOFRASTERRECORD );

			for ( addressNum = 0; addressNum < NUMOFADDRESSRECORDSPERCORE; addressNum++ ) {

			    currentAddress = addressBase + ( addressNum * SIZEOFADDRESSRECORD );
			
			}
	    }
	}
}
