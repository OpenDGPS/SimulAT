
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUMOFRASTERRECORDSPERCORE 3 // 160  // defined by num of raster records ~80k divided by num of GPU cores ~512

// rasters are stored in  int(4Byte): rasterDd, int(4Byte): minLat, int(4Byte): minLon, int(4Byte): maxLat, int(4Byte): maxLon, int(4Byte): [empty]
#define SIZEOFRASTERRECORD 5 // DWORDS to jump between the records

#define NUMOFADDRESSRECORDSPERBLOCK 4 // 5000 // defined by num of address records ~2.5m divided by num of GPU cores ~512
#define NUMOFADDRESSBLOCKS 3 // 512 // equal to the number of GPU cores

// addresses are stored in int(4Byte): id, int(4Byte): lat, int(4Byte): lon, int(4Byte): [rasterId]
#define SIZEOFADDRESSRECORD 4 // DWORDS to jump between the records

int globalThreadId = 0;

// __global__ 

void mapRasterToAddresses(int rasterBase, int addressRecords) {

	int threadId, recordNum, addressBlockNum, currentAddressBlockNum, addressBase, addressNum, currentRasterAddress, currentAddressAddress;
	
	threadId = globalThreadId;

	for ( addressBlockNum = 0; addressBlockNum < NUMOFADDRESSBLOCKS; addressBlockNum++ ) {
	
		currentAddressBlockNum = ( addressBlockNum + threadId ) % NUMOFADDRESSBLOCKS;
	    
	    addressBase = addressRecords + ( currentAddressBlockNum * NUMOFADDRESSRECORDSPERBLOCK * SIZEOFADDRESSRECORD );

		for ( recordNum = 0; recordNum < NUMOFRASTERRECORDSPERCORE; recordNum++ ) {

			currentRasterAddress = rasterBase + ( recordNum * SIZEOFRASTERRECORD ) + ( threadId * SIZEOFRASTERRECORD );

			for ( addressNum = 0; addressNum < NUMOFADDRESSRECORDSPERBLOCK; addressNum++ ) {

				currentAddressAddress = addressBase + ( addressNum * SIZEOFADDRESSRECORD );
				
				printf("threadId:\t%d\n",threadId);

				printf("\taddressBlockNum:\t%d\tcurrentAddressBlockNum:\t%d\taddressBase:\t%d\n",addressBlockNum,currentAddressBlockNum,addressBase);

				printf("\t\trecordNum:\t%d\tcurrentRasterAddress:\t%d\n",recordNum,currentRasterAddress);

				printf("\t\t\taddressNum = %d \tcurrentAddressAddress = %d\n", addressNum, currentAddressAddress);

				printf("if currentRasterAddress:[%d][1] < currentAddressAddress:[%d][1]\n", currentRasterAddress, currentAddressAddress);
			
			}
	    }
	}
}
