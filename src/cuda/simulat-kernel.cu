
#include <hip/hip_runtime.h>

#define NUM_OF_PERSONS_FIELD_IN_RASTER 4

//  int genderLUT = [0,0,0,0,0,0,0,0,0,0,1,1,1,1,1,1,1,1,1];
//  int ageMinLUT = [0,3,6,9,13,18,24,34,44,54,69,79,0,3,6,9,13,18,24,34,44,54,69,79];
//  int ageMaxLut = [2,5,8,12,17,23,33,43,53,68,78,2,5,8,12,17,23,33,43,53,68,78];

#define NUMOFRASTERRECORDSPERCORE 160  // defined by num of raster records ~80k divided by num of GPU cores ~512
#define SIZEOFRASTERRECORD 4 // DWORDS to jump between the records

#define NUMOFADDRESSRECORDSPERCORE 5000 // defined by num of address records ~2.5m divided by num of GPU cores ~512
#define NUMOFADDRESSBLOCKS 512 // equal to the number of GPU cores
#define SIZEOFADDRESSRECORD 5 // DWORDS to jump between the records

__global__ void mapRasterToAddresses(int rasterRecords, int addressRecords) {

	int threadId, recordNum, addressBlockNum, currentAddressBlockNum, addressNumInBlock, rasterBase, addressBase, addressNum, currentRaster, currentAddress;
	
	for ( addressBlockNum = 0; addressBlockNum < NUMOFADDRESSBLOCKS; addressBlockNum++ ) {
	
		currentAddressBlockNum = ( addressBlockNum + threadId ) % NUMOFADDRESSBLOCKS;
	    
	    addressBase = addressRecords + ( currentAddressBlockNum * NUMOFADDRESSRECORDSPERCORE * SIZEOFADDRESSRECORD );

	    for ( recordNum = 0; recordNum < NUMOFRASTERRECORDSPERCORE; recordNum++ ) {

	    	currentRaster = rasterRecords + ( recordNum * SIZEOFRASTERRECORD ) + ( threadId * SIZEOFRASTERRECORD );

			for ( addressNum = 0; addressNum < NUMOFADDRESSRECORDSPERCORE; addressNum++ ) {

			    currentAddress = addressBase + ( addressNum * SIZEOFADDRESSRECORD );
			
			}
	    }
	}
}

/*
__device__ void createPersonsFromRaster(int *rasterRecord) {
	   int numOfPersons, numOfHouseholds, pickAdult, i, j, k;
	   numOfPersons = rasterRecord[NUM_OF_PERSONS_FIELD_IN_RASTER];
	   int listOfFieldsWithAvailableAdults = [0,1,2,3,4,5,6,7,8,9,10]; // needs to be initialized
	   int listOfFieldsWithAvailableHH = [0,1,2,3,4];
	   for ( i = 0; i < numOfPersons; i++ ) {
	       pickAdult = random(0 to length(listOfFieldsWithAvailableAdults));
	       personId = i;
	       gender = genderLUT[listOfFieldsWithAvailableAdults[pickAdult]];
	       ageMin = ageMinLUT[listOfFieldsWithAvailableAdults[pickAdult]];
	       ageMax = ageMaxLUT[listOfFieldsWithAvailableAdults[pickAdult]];
	       age = random(ageMin,ageMax);
	       
	       rasterRecord[NUM_OF_PERSONS_FIELD_IN_RASTER + listOfFieldsWithAvailableAdults[pickAdult]]--;
	       if ( rasterRecord[NUM_OF_PERSONS_FIELD_IN_RASTER + listOfFieldsWithAvailableAdults[pickAdult]] == 0 ) {
	       	  remove pickAdult from listOfFieldsWithAvailableAdults;
		}
		if ( availableInHH == 0 ) {
		   pickHH = random(0 to length(listOfFieldsWithAvailableHH));
		   rasterRecord[HHID_START_AT + pickHH]--;
		   if ( listOfFieldsWithAvailableHH[piHH] == 3 ) // 3 to 5 persons
		      switch ( random (0 to 100) < 50 )
		      	     case < 50:
			     	  hhSize = 3;
			     case < 85:
			     	  hhSize = 4;
			     default:
				hhSize = 5;
		   if ( listOfFieldsWithAvailableHH[piHH] == 4 ) // 6+ persons {
                      switch ( random (0 to 100) < 50 )
		            	case < 30:
                        	hhSize = 6;
			 			case < 55:
                            hhSize = 7;
                        case < 65:
                            hhSize = 8;
                        case < 75:
                            hhSize = 9;
                        case < 85:
                            hhSize = 10;
                        case < 92:
                            hhSize = 11;
                        case < 95:
                            hhSize = 12;
                        case < 98:
                            hhSize = 13;
                        default:
                        	hhSize = random (13 to 25);
			}		   
		}
		  	  
		  
		  
	   }
}
*/