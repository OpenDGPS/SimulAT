
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUMOFRASTERRECORDSPERCORE 3 // 160  // defined by num of raster records ~80k divided by num of GPU cores ~512

// rasters are stored in  int(4Byte): rasterDd, int(4Byte): minLat, int(4Byte): minLon, int(4Byte): maxLat, int(4Byte): maxLon, int(4Byte): [empty]
#define SIZEOFRASTERRECORD 5 // DWORDS to jump between the records

#define NUMOFADDRESSRECORDSPERBLOCK 4 // 5000 // defined by num of address records ~2.5m divided by num of GPU cores ~512
#define NUMOFADDRESSBLOCKS 3 // 512 // equal to the number of GPU cores

// addresses are stored in int(4Byte): id, int(4Byte): lat, int(4Byte): lon, int(4Byte): [rasterId]
#define SIZEOFADDRESSRECORD 4 // DWORDS to jump between the records

#define ADDLAT 1
#define ADDLON 2
#define MINLAT 1
#define MINLON 2
#define MAXLAT 3
#define MAXLON 4

int globalThreadId = 0;

__global__ void saxpy(int n, float a, float *x, float *y) {

	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n) y[i] = a*x[i] + y[i];
}
// __global__ 

void mapRasterToAddresses(int rasterBase, int addressRecords) {

	int threadId, recordNum, addressBlockNum, currentAddressBlockNum, addressBase, addressNum, currentRasterAddress, currentAddressAddress;
	
	// int rMinLat, rMaxLat, rMinLon, rMaxLon, aLat, aLon;

	threadId = globalThreadId;

	for ( addressBlockNum = 0; addressBlockNum < NUMOFADDRESSBLOCKS; addressBlockNum++ ) {
	
		currentAddressBlockNum = ( addressBlockNum + threadId ) % NUMOFADDRESSBLOCKS;
	    
	    addressBase = addressRecords + ( currentAddressBlockNum * NUMOFADDRESSRECORDSPERBLOCK * SIZEOFADDRESSRECORD );

		for ( recordNum = 0; recordNum < NUMOFRASTERRECORDSPERCORE; recordNum++ ) {

			currentRasterAddress = rasterBase + ( recordNum * SIZEOFRASTERRECORD ) + ( threadId * SIZEOFRASTERRECORD );

			for ( addressNum = 0; addressNum < NUMOFADDRESSRECORDSPERBLOCK; addressNum++ ) {

				currentAddressAddress = addressBase + ( addressNum * SIZEOFADDRESSRECORD );
				
				printf("threadId:\t%d\n",threadId);

				printf("\taddressBlockNum:\t%d\tcurrentAddressBlockNum:\t%d\taddressBase:\t%d\n",addressBlockNum,currentAddressBlockNum,addressBase);

				printf("\t\trecordNum:\t%d\tcurrentRasterAddress:\t%d\n",recordNum,currentRasterAddress);

				printf("\t\t\taddressNum = %d \tcurrentAddressAddress = %d\n", addressNum, currentAddressAddress);

				/*
				rMinLat = currentRasterAddress[currentRasterAddress][MINLAT];
				rMaxLat = currentRasterAddress[currentRasterAddress][MAXLAT];
				rMinLon = currentRasterAddress[currentRasterAddress][MINLON];
				rMaxLon = currentRasterAddress[currentRasterAddress][MAXLON];
				aLat = currentAddressAddress[currentAddressAddress][ADDLAT];
				aLon = currentAddressAddress[currentAddressAddress][ADDLON];
				printf("rMinLat > aLat\n");
				printf("rMaxLat < aLat\n");
				printf("rMinLon > aLon\n");
				printf("rMaxLon > aLon\n");
				*/
			
			}
	    }
	}
}
