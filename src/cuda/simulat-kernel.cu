
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUMOFRASTERRECORDSPERCORE 16 // 160  // defined by num of raster records ~80k divided by num of GPU cores ~512
#define SIZEOFRASTERRECORD 4 // DWORDS to jump between the records

#define NUMOFADDRESSRECORDSPERCORE 5 // 5000 // defined by num of address records ~2.5m divided by num of GPU cores ~512
#define NUMOFADDRESSBLOCKS 12 // 512 // equal to the number of GPU cores
#define SIZEOFADDRESSRECORD 5 // DWORDS to jump between the records

int globalThreadId = 0;

// __global__ 

void mapRasterToAddresses(int rasterRecords, int addressRecords) {

	int threadId, recordNum, addressBlockNum, currentAddressBlockNum, addressNumInBlock, rasterBase, addressBase, addressNum, currentRasterAddress, currentAddressAddress;
	
	threadId = globalThreadId;

	printf("threadId:\t%d\n",threadId);

	for ( addressBlockNum = 0; addressBlockNum < NUMOFADDRESSBLOCKS; addressBlockNum++ ) {
	
		currentAddressBlockNum = ( addressBlockNum + threadId ) % NUMOFADDRESSBLOCKS;
	    
	    addressBase = addressRecords + ( currentAddressBlockNum * NUMOFADDRESSRECORDSPERCORE * SIZEOFADDRESSRECORD );

		printf("\taddressBlockNum:\t%d\tcurrentAddressBlockNum:\t%d\taddressBase:\t%d\n",addressBlockNum,currentAddressBlockNum,addressBase);

		for ( recordNum = 0; recordNum < NUMOFRASTERRECORDSPERCORE; recordNum++ ) {

			currentRasterAddress = rasterRecords + ( recordNum * SIZEOFRASTERRECORD ) + ( threadId * SIZEOFRASTERRECORD );

			printf("\t\t\trecordNum:\t%d\n",recordNum);

			for ( addressNum = 0; addressNum < NUMOFADDRESSRECORDSPERCORE; addressNum++ ) {

				currentAddressAddress = addressBase + ( addressNum * SIZEOFADDRESSRECORD );
				
				printf("threadId = %d \taddressBase = %d \tcurrentRaster = %d \tcurrentAddress = %d\n",threadId, addressBase, currentRasterAddress,currentAddressAddress);
			
			}
	    }
	}
}
