
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUMOFRASTERRECORDSPERCORE 160  // defined by num of raster records ~80k divided by num of GPU cores ~512
#define SIZEOFRASTERRECORD 4 // DWORDS to jump between the records

#define NUMOFADDRESSRECORDSPERCORE 5000 // defined by num of address records ~2.5m divided by num of GPU cores ~512
#define NUMOFADDRESSBLOCKS 512 // equal to the number of GPU cores
#define SIZEOFADDRESSRECORD 5 // DWORDS to jump between the records

// __global__ 

void mapRasterToAddresses(int rasterRecords, int addressRecords) {

	int threadId, recordNum, addressBlockNum, currentAddressBlockNum, addressNumInBlock, rasterBase, addressBase, addressNum, currentRaster, currentAddress;
	
	threadId = 1;

	for ( addressBlockNum = 0; addressBlockNum < NUMOFADDRESSBLOCKS; addressBlockNum++ ) {
	
		currentAddressBlockNum = ( addressBlockNum + threadId ) % NUMOFADDRESSBLOCKS;
	    
	    addressBase = addressRecords + ( currentAddressBlockNum * NUMOFADDRESSRECORDSPERCORE * SIZEOFADDRESSRECORD );

	    for ( recordNum = 0; recordNum < NUMOFRASTERRECORDSPERCORE; recordNum++ ) {

			currentRaster = rasterRecords + ( recordNum * SIZEOFRASTERRECORD ) + ( threadId * SIZEOFRASTERRECORD );

			for ( addressNum = 0; addressNum < NUMOFADDRESSRECORDSPERCORE; addressNum++ ) {

				currentAddress = addressBase + ( addressNum * SIZEOFADDRESSRECORD );
				
				printf("threadId = %d \taddressBase = %d\n",threadId, addressBase);
			
			}
	    }
	}
}
