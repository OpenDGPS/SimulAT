
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUMOFRASTERRECORDSPERCORE 3 // 160  // defined by num of raster records ~80k divided by num of GPU cores ~512
#define SIZEOFRASTERRECORD 4 // DWORDS to jump between the records

#define NUMOFADDRESSRECORDSPERCORE 4 // 5000 // defined by num of address records ~2.5m divided by num of GPU cores ~512
#define NUMOFADDRESSBLOCKS 3 // 512 // equal to the number of GPU cores
#define SIZEOFADDRESSRECORD 5 // DWORDS to jump between the records

int globalThreadId = 0;

// __global__ 

void mapRasterToAddresses(int rasterRecords, int addressRecords) {

	int threadId, recordNum, addressBlockNum, currentAddressBlockNum, addressNumInBlock, rasterBase, addressBase, addressNum, currentRasterAddress, currentAddressAddress;
	
	threadId = globalThreadId;

	printf("threadId:\t%d\n",threadId);

	for ( addressBlockNum = 0; addressBlockNum < NUMOFADDRESSBLOCKS; addressBlockNum++ ) {
	
		currentAddressBlockNum = ( addressBlockNum + threadId ) % NUMOFADDRESSBLOCKS;
	    
	    addressBase = addressRecords + ( currentAddressBlockNum * NUMOFADDRESSRECORDSPERCORE * SIZEOFADDRESSRECORD );

		printf("\taddressBlockNum:\t%d\tcurrentAddressBlockNum:\t%d\taddressBase:\t%d\n",addressBlockNum,currentAddressBlockNum,addressBase);

		for ( recordNum = 0; recordNum < NUMOFRASTERRECORDSPERCORE; recordNum++ ) {

			currentRasterAddress = rasterRecords + ( recordNum * SIZEOFRASTERRECORD ) + ( threadId * SIZEOFRASTERRECORD );

			printf("\t\t\trecordNum:\t%d\tcurrentRasterAddress:\t%d\n",recordNum,currentRasterAddress);

			for ( addressNum = 0; addressNum < NUMOFADDRESSRECORDSPERCORE; addressNum++ ) {

				currentAddressAddress = addressBase + ( addressNum * SIZEOFADDRESSRECORD );
				
				printf("\t\t\t\taddressNum = %d \tcurrentAddressAddress = %d\n", addressNum, currentAddressAddress);
			
			}
	    }
	}
}
