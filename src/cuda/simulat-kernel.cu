
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUMOFRASTERRECORDSPERCORE 3 // 160  // defined by num of raster records ~80k divided by num of GPU cores ~512

// rasters are stored in  int(4Byte): rasterDd, int(4Byte): minLat, int(4Byte): minLon, int(4Byte): maxLat, int(4Byte): maxLon, int(4Byte): [empty]
#define SIZEOFRASTERRECORD 5 // DWORDS to jump between the records

#define NUMOFADDRESSRECORDSPERBLOCK 4 // 5000 // defined by num of address records ~2.5m divided by num of GPU cores ~512
#define NUMOFADDRESSBLOCKS 3 // 512 // equal to the number of GPU cores

// addresses are stored in int(4Byte): id, int(4Byte): lat, int(4Byte): lon, int(4Byte): [rasterId]
#define SIZEOFADDRESSRECORD 4 // DWORDS to jump between the records

#define ADDLAT 1
#define ADDLON 2
#define MINLAT 1
#define MINLON 2
#define MAXLAT 3
#define MAXLON 4

int globalThreadId = 0;

// __global__ 

void mapRasterToAddresses(int rasterBase, int addressRecords) {

	int threadId, recordNum, addressBlockNum, currentAddressBlockNum, addressBase, addressNum, currentRasterAddress, currentAddressAddress;
	
	threadId = globalThreadId;

	for ( addressBlockNum = 0; addressBlockNum < NUMOFADDRESSBLOCKS; addressBlockNum++ ) {
	
		currentAddressBlockNum = ( addressBlockNum + threadId ) % NUMOFADDRESSBLOCKS;
	    
	    addressBase = addressRecords + ( currentAddressBlockNum * NUMOFADDRESSRECORDSPERBLOCK * SIZEOFADDRESSRECORD );

		for ( recordNum = 0; recordNum < NUMOFRASTERRECORDSPERCORE; recordNum++ ) {

			currentRasterAddress = rasterBase + ( recordNum * SIZEOFRASTERRECORD ) + ( threadId * SIZEOFRASTERRECORD );

			for ( addressNum = 0; addressNum < NUMOFADDRESSRECORDSPERBLOCK; addressNum++ ) {

				currentAddressAddress = addressBase + ( addressNum * SIZEOFADDRESSRECORD );
				
				printf("threadId:\t%d\n",threadId);

				printf("\taddressBlockNum:\t%d\tcurrentAddressBlockNum:\t%d\taddressBase:\t%d\n",addressBlockNum,currentAddressBlockNum,addressBase);

				printf("\t\trecordNum:\t%d\tcurrentRasterAddress:\t%d\n",recordNum,currentRasterAddress);

				printf("\t\t\taddressNum = %d \tcurrentAddressAddress = %d\n", addressNum, currentAddressAddress);

				printf("if currentRasterAddress:[%d][MINLAT] > currentAddressAddress:[%d][ADDLAT]\n", currentRasterAddress, currentAddressAddress);
				printf("if currentRasterAddress:[%d][MAXLAT] < currentAddressAddress:[%d][ADDLAT]\n", currentRasterAddress, currentAddressAddress);
				printf("if currentRasterAddress:[%d][MINLON] > currentAddressAddress:[%d][ADDLON]\n", currentRasterAddress, currentAddressAddress);
				printf("if currentRasterAddress:[%d][MAXLON] > currentAddressAddress:[%d][ADDLON]\n", currentRasterAddress, currentAddressAddress);
			
			}
	    }
	}
}
