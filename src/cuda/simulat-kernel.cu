
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUMOFRASTERRECORDSPERCORE 3 // 160  // defined by num of raster records ~80k divided by num of GPU cores ~512

// rasters are stored in  int(4Byte): rasterDd, int(4Byte): lat, int(4Byte): lon, int(4Byte): [empty]
#define SIZEOFRASTERRECORD 4 // DWORDS to jump between the records

#define NUMOFADDRESSRECORDSPERCORE 4 // 5000 // defined by num of address records ~2.5m divided by num of GPU cores ~512
#define NUMOFADDRESSBLOCKS 3 // 512 // equal to the number of GPU cores

// addresses are stored in int(4Byte): id, int(4Byte): lat, int(4Byte): lon, int(4Byte): [rasterId]
#define SIZEOFADDRESSRECORD 5 // DWORDS to jump between the records

int globalThreadId = 0;

// __global__ 

void mapRasterToAddresses(int rasterBase, int addressRecords) {

	int threadId, recordNum, addressBlockNum, currentAddressBlockNum, addressNumInBlock, addressBase, addressNum, currentRasterAddress, currentAddressAddress;
	
	threadId = globalThreadId;

	for ( addressBlockNum = 0; addressBlockNum < NUMOFADDRESSBLOCKS; addressBlockNum++ ) {
	
		currentAddressBlockNum = ( addressBlockNum + threadId ) % NUMOFADDRESSBLOCKS;
	    
	    addressBase = addressRecords + ( currentAddressBlockNum * NUMOFADDRESSRECORDSPERCORE * SIZEOFADDRESSRECORD );

		for ( recordNum = 0; recordNum < NUMOFRASTERRECORDSPERCORE; recordNum++ ) {

			currentRasterAddress = rasterBase + ( recordNum * SIZEOFRASTERRECORD ) + ( threadId * SIZEOFRASTERRECORD );

			for ( addressNum = 0; addressNum < NUMOFADDRESSRECORDSPERCORE; addressNum++ ) {

				currentAddressAddress = addressBase + ( addressNum * SIZEOFADDRESSRECORD );
				
				printf("threadId:\t%d\n",threadId);

				printf("\taddressBlockNum:\t%d\tcurrentAddressBlockNum:\t%d\taddressBase:\t%d\n",addressBlockNum,currentAddressBlockNum,addressBase);

				printf("\t\trecordNum:\t%d\tcurrentRasterAddress:\t%d\n",recordNum,currentRasterAddress);

				printf("\t\t\taddressNum = %d \tcurrentAddressAddress = %d\n", addressNum, currentAddressAddress);
			
			}
	    }
	}
}
